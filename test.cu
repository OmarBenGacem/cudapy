#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <testhead.h>


int main() {

    int a[] = {1, 2, 3};
    int b[] = {4, 5, 3};
    int c[sizeof(a) / sizeof(int)] = {0};

    int* cudaA = 0;
    int* cudaB = 0;
    int* cudaC = 0;

    hipMalloc(&cudaA, sizeof(a));
    hipMalloc(&cudaB, sizeof(b));
    hipMalloc(&cudaC, sizeof(c));

    hipMemcpy(cudaA, a, sizeof(a), hipMemcpyHostToDevice);
    hipMemcpy(cudaB, b, sizeof(b), hipMemcpyHostToDevice);
    
    V_Add_and_Two<<< 1, sizeof(a) / sizeof(int)>>>(cudaA, cudaB, cudaC);

    hipMemcpy(c, cudaC, sizeof(c), hipMemcpyDeviceToHost);
    printf("{1,2,3} + {4,5,3} = {%d,%d,%d}\n",
        c[0], c[1], c[2]);

    hipFree(cudaA);
    hipFree(cudaB);
    hipFree(cudaC);

    return 0;
}